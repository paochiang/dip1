#include "hip/hip_runtime.h"
/*
Copyright (c) 2013-2015, Gregory P. Meyer
                         University of Illinois Board of Trustees
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright
      notice, this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of the copyright holder(s) nor the names of its
      contributors may be used to endorse or promote products derived from
      this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <dip/common/error.h>
#include <dip/common/types.h>

#define BLOCK_WIDTH 16

namespace dip {

__global__ void ThresholdFilter(int min_depth, int max_depth,
                                int width, int height, Depth *depth) {
  // Get Block and Thread Id
  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  // Calculate Row & Column
  int col = tx + bx * BLOCK_WIDTH;
  int row = ty + by * BLOCK_WIDTH;

  // Perform Threshold
  if ((col < width) && (row < height)) {
    int i = col + row * width;

    int depth_value = depth[i];

    if((depth_value < min_depth) || (depth_value > max_depth))
      depth[i] = 0;
  }
}

void ThresholdKernel(int min_depth, int max_depth, int width, int height,
                     Depth *depth) {
  // Launch Threshold Filter Kernel
  int grid_width = (width + (BLOCK_WIDTH - 1)) / BLOCK_WIDTH;
  int grid_height = (height + (BLOCK_WIDTH - 1)) / BLOCK_WIDTH;

  dim3 grid_dim(grid_width, grid_height, 1);
  dim3 block_dim(BLOCK_WIDTH, BLOCK_WIDTH, 1);

  ThresholdFilter<<<grid_dim, block_dim>>>(min_depth, max_depth, width, height,
                                           depth);

  CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

} // namespace dip
